#include "hip/hip_runtime.h"
#include "fftFilter.h"

__global__ void warmUpKernel(float* d_src1, float* d_src2, float* d_dst, int nx, int ny) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= nx || y >= ny) return;
    d_dst[y * nx + x] = d_src1[y * nx + x] + d_src2[y * nx + x];
}

void warmUpGPU() {
    const int x = 1024;
    const int y = 1024;
    float* h_src1 = new float[x * y];
    float* h_src2 = new float[x * y];
    float* h_dst = new float[x * y];
    float* d_src1;
    float* d_src2;
    float* d_dst;
    Check(hipMalloc(&d_src1, x * y * sizeof(float)));
    Check(hipMalloc(&d_src2, x * y * sizeof(float)));
    Check(hipMalloc(&d_dst, x * y * sizeof(float)));

    dim3 block(32, 32);
    dim3 grid(iDivUp(x, 32), iDivUp(y, 32));

    Check(hipMemcpy(d_src1, h_src1, x * y * sizeof(float), hipMemcpyHostToDevice));
    Check(hipMemcpy(d_src2, h_src2, x * y * sizeof(float), hipMemcpyHostToDevice));

    warmUpKernel << <grid, block >> > (d_src1, d_src2, d_dst, x, y);

    Check(hipMemcpy(h_dst, d_dst, x * y * sizeof(float), hipMemcpyDeviceToHost));

    Check(hipFree(d_src1));
    Check(hipFree(d_src2));
    Check(hipFree(d_dst));
    delete[] h_src1;
    delete[] h_src2;
    delete[] h_dst;
}
